//
// Created by xuming on 21-6-18.
//
//#include <iomanip>

#include <hip/hip_runtime.h>
#include <iostream>
//#include <stdio.h>
//
//#include <cuda.h>
//#include <cuda_runtime.h>
//#include <device_launch_parameters.h>

using namespace std;

//
////检测GPU
//bool CheckCUDA(void){
//    int count = 0;
//    int i = 0;
//
//    cudaGetDeviceCount(&count);
//    if (count == 0) {
//        printf("找不到支持CUDA的设备!\n");
//        return false;
//    }
//    cudaDeviceProp prop;
//    for (i = 0; i < count; i++) {
//        if (cudaGetDeviceProperties(&prop, i) == cudaSuccess) {
//            if (prop.major >= 1) {
//                break;
//            }
//        }
//    }
//    if (i == count) {
//        printf("找不到支持CUDA的设备!\n");
//        return false;
//    }
//    cudaGetDeviceProperties(&prop, 0);
//    printf("GPU is: %s\n", prop.name);
//    cudaSetDevice(0);
//    printf("CUDA initialized success.\n");
//    return true;
//}


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


int main()
{
    hipDeviceProp_t prop;
    int dev;

    HANDLE_ERROR(hipGetDevice(&dev));
    printf("ID of current CUDA device: %d\n", dev);

    memset( &prop, 0, sizeof(hipDeviceProp_t));


    prop.major = 1;
    prop.minor = 6;
    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );
    printf( "ID of CUDA device closest to revision 1.3:  %d\n", dev );

    HANDLE_ERROR( hipSetDevice( dev ) );




}