#include <iomanip>
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>



using namespace std;


//检测GPU
bool CheckCUDA(void){
    int count = 0;
    int i = 0;

    hipGetDeviceCount(&count);
    if (count == 0) {
        printf("找不到支持CUDA的设备!\n");
        return false;
    }
    hipDeviceProp_t prop;
    for (i = 0; i < count; i++) {
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }
    if (i == count) {
        printf("找不到支持CUDA的设备!\n");
        return false;
    }
    hipGetDeviceProperties(&prop, 0);
    printf("GPU is: %s\n", prop.name);
    hipSetDevice(0);
    printf("CUDA initialized success.\n");
    return true;
}



__global__ void kernel(void){

}


int main( void ) {
    kernel <<<1, 1>>>();
    printf( "Hello, World!\n" );
    return 0;
}

/*
 * 一个空的函数 kernel(), 并且带有修饰符 __global__
 * 对这个空函数的调用，并且带有修饰字符 <<<1, 1>>>
 *
 * CUDA C 为标准 C 增加 __global__ 修饰符，这个修饰符告诉编译器，函数应该运行在设备上而不是主机上运行.
 */