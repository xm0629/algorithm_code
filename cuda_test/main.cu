#include <iomanip>
#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>



using namespace std;


//检测GPU
bool CheckCUDA(void){
    int count = 0;
    int i = 0;

    hipGetDeviceCount(&count);
    if (count == 0) {
        printf("找不到支持CUDA的设备!\n");
        return false;
    }
    hipDeviceProp_t prop;
    for (i = 0; i < count; i++) {
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }
    if (i == count) {
        printf("找不到支持CUDA的设备!\n");
        return false;
    }
    hipGetDeviceProperties(&prop, 0);
    printf("GPU is: %s\n", prop.name);
    hipSetDevice(0);
    printf("CUDA initialized success.\n");
    return true;
}


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))



#define COUNT(M) M * M               //定义有参宏


int main( void ) {
    printf( "Hello, World!\n" );


    int x = 6;
    printf("COUNT = %d\n", COUNT(x + 1));// 输出结果： COUNT = 13
    printf("COUNT = %d\n", COUNT(++x));  // 输出结果： COUNT = 56


    return 0;
}
