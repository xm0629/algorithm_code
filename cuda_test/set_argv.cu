#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "utils.h"


__global__ void add(int a, int b, int *c){
    *c = a + b;
}


int main()
{
    int c;
    int *dev_c;

    HANDLE_ERROR( hipMalloc( (void**)&dev_c, sizeof(int) ) );

    add<<<1, 1>>>(2, 7, dev_c);
    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

    std::cout << "2 + 7 = " << c << std::endl;

    hipFree(dev_c);

    return 0;
}

/*
 * 可以像调用 C 函数一样传递参数
 * 当设备执行任何有用操作时，都需要分配内存，例如将计算值返回给主机
 * hipMalloc() 来分配内存，这个函数调用的行为类似于标准 c 的 malloc() 但该函数的作用是告诉 cuda 运行时在设备上分配的内存,
 * 第一个参数是一个指针，指向用于保存新分配内存地址的变量， 第二个参数是分配内存的大小, 除了分配内存的指针不是作为函数的返回值外, 这个函数的行为与 malloc 都是相同的.
 *
 */