#include <iostream>
#include <hip/hip_runtime.h>




static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))




__device__ int addem(int a, int b){
    return a + b;
}


__global__ void add(int a, int b, int *c){
    *c = addem(a, b);

}



int main()
{
    int c;
    int *dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int) ) );
    add<<<1,1>>>( 2, 7, dev_c );

    HANDLE_ERROR( hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost) );
    printf( "2 + 7 = %d\n", c );
    HANDLE_ERROR( hipFree( dev_c ) );
    return 0;
}